#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <bitset>
#include <chrono>
#include <stdexcept>
#include "lodepng.h"
#include "LbpSeq.h"
#include "LbpPar.h"
#include "LbpCuda.h"
#include "Benchmark.h"

static std::vector<byte> pixels;
static unsigned width;
static unsigned height;

static bool loadImage(const std::string& filename)
{
	unsigned error = lodepng::decode(pixels, width, height, filename);
	if (error) {
		std::cerr << lodepng_error_text(error) << std::endl;
		return false;
	}
	std::cout << "image size is " << width << "x" << height << std::endl;
	return true;
}

static void makeSampleOutput()
{
	LbpSeq output(pixels, width, height);
	LbpPar output2(pixels, width, height);
	LbpCuda output3(pixels, width, height);

	std::vector<byte> outputImage;
	unsigned outputWidth, outputHeight;
	output.calculateNormalizedLBPs(2, 8, 32, &outputImage, &outputWidth, &outputHeight);

	std::string outI;
	std::cout << "Insert Path for output" << std::endl;
	std::getline(std::cin, outI);

	lodepng::encode(outI, outputImage, outputWidth, outputHeight, LCT_GREY);

}



static void test()
{
	LbpSeq image(pixels, width, height);
	LbpPar d_Image(pixels, width, height);
	LbpCuda d_Image_cuda(pixels, width, height);
	int samp[] = { 4, 6, 8, 9, 10, 11, 12, 13, 14, 15 };
	float rads[] = { 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0, 2.0 };
	//int edge[] = { 16, 16, 16, 16, 16, 16, 16, 16, 16, 16 };
	int edge[] = { 32, 32, 32, 32, 32, 32, 32, 32, 32, 32 };
	for (int i = 0; i < 10; i++) {
		long cpuMillis[1000] = {};
		long totCPU = 0;
		long gpuMillis[1000] = {};
		long totGPU = 0;
		long gpuMillisCUDA[1000] = {};
		long totCUDA = 0;
		try {
			for (int j = 0; j < 1000; j++) {
				Benchmark::start();
				auto cpuHistograms = image.calculateNormalizedLBPs(rads[i], samp[i], edge[i]);
				Benchmark::stop();
				cpuMillis[j] = Benchmark::getMillis();
				totCPU += cpuMillis[j];
				Benchmark::start();
				auto gpuHistograms = d_Image.calculateNormalizedLBPs(rads[i], samp[i], edge[i]);
				Benchmark::stop();
				gpuMillis[j] = Benchmark::getMillis();
				totGPU += gpuMillis[j];
				Benchmark::start();
				auto gpuHistograms_CUDA = d_Image_cuda.calculateNormalizedLBPs(rads[i], samp[i], edge[i]);
				Benchmark::stop();
				gpuMillisCUDA[j] = Benchmark::getMillis();
				totCUDA += gpuMillisCUDA[j];
				std::cerr << "With conf {s=" << samp[j] << "; r=" << rads[j] << "; e=" << edge[j] << "} ";
				std::cerr << "\tSequential took " << totCPU / 1000 << "ms and Parallel " << totGPU / 1000 << "ms and Cuda " << totCUDA / 1000 << "ms";
			}
		}

		catch (const std::invalid_argument& e) {
			std::cerr << e.what() << std::endl;
			std::cerr << "Conf {s=" << samp[i] << "; r=" << rads[i] << "; e=" << edge[i] << "} is not supported" << std::endl;
			std::cerr << "End of main" << std::endl;
			return;
		}
	}
}

int main()
{
	std::string s;
	std::cout << "Insert Path" << std::endl;
	std::getline(std::cin, s);
	bool t = loadImage(s);
	makeSampleOutput();
	//test();
	hipDeviceReset();
	return 0;
}

